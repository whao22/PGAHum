#include <hip/hip_runtime.h>

#include <iostream>
#define CUDA_CHECK(condition) \
  /* Code block avoids redefinition of cudaError_t error */ \
  do { \
    hipError_t error = condition; \
    if(error!=hipSuccess) \
    { \
    	std::cerr<<"Error: "<<hipGetErrorString(error)<<std::endl; \
    	exit(1); \
    } \
  } while (0)

#define CUDA_POST_KERNEL_CHECK hipDeviceSynchronize();CUDA_CHECK(hipPeekAtLastError())

__inline__ __device__ int index(int r, int c)
{
	return 4*r+c;
}
template <typename scalar_t>
__global__ void cu4x4MInv(const scalar_t* ms, scalar_t* invs, bool* checks, int N)
{
	int mid=threadIdx.x+blockIdx.x*blockDim.x;
	if(mid>=N)
		return;
	scalar_t* inv=invs+16*mid;
	const scalar_t* m=ms+16*mid;
	scalar_t cof00=m[index(1,1)]*(m[index(3,3)]*m[index(2,2)]-m[index(2,3)]*m[index(3,2)]) -m[index(1,2)]*(m[index(3,3)]*m[index(2,1)]-m[index(3,1)]*m[index(2,3)]) +m[index(1,3)]*(m[index(3,2)]*m[index(2,1)]-m[index(2,2)]*m[index(3,1)]);
	scalar_t cof01=-(m[index(1,0)]*(m[index(3,3)]*m[index(2,2)]-m[index(2,3)]*m[index(3,2)]) -m[index(1,2)]*(m[index(3,3)]*m[index(2,0)]-m[index(3,0)]*m[index(2,3)]) +m[index(1,3)]*(m[index(3,2)]*m[index(2,0)]-m[index(2,2)]*m[index(3,0)]));
	scalar_t cof02=m[index(1,0)]*(m[index(3,3)]*m[index(2,1)]-m[index(2,3)]*m[index(3,1)]) -m[index(1,1)]*(m[index(3,3)]*m[index(2,0)]-m[index(3,0)]*m[index(2,3)]) +m[index(1,3)]*(m[index(3,1)]*m[index(2,0)]-m[index(2,1)]*m[index(3,0)]);
	scalar_t cof03=-(m[index(1,0)]*(m[index(3,2)]*m[index(2,1)]-m[index(2,2)]*m[index(3,1)]) -m[index(1,1)]*(m[index(3,2)]*m[index(2,0)]-m[index(3,0)]*m[index(2,2)]) +m[index(1,2)]*(m[index(3,1)]*m[index(2,0)]-m[index(2,1)]*m[index(3,0)]));
	
	scalar_t cof10=-(m[index(0,1)]*(m[index(3,3)]*m[index(2,2)]-m[index(2,3)]*m[index(3,2)]) -m[index(0,2)]*(m[index(3,3)]*m[index(2,1)]-m[index(3,1)]*m[index(2,3)]) +m[index(0,3)]*(m[index(3,2)]*m[index(2,1)]-m[index(2,2)]*m[index(3,1)]));
	scalar_t cof11=m[index(0,0)]*(m[index(3,3)]*m[index(2,2)]-m[index(2,3)]*m[index(3,2)]) -m[index(0,2)]*(m[index(3,3)]*m[index(2,0)]-m[index(3,0)]*m[index(2,3)]) +m[index(0,3)]*(m[index(3,2)]*m[index(2,0)]-m[index(2,2)]*m[index(3,0)]);
	scalar_t cof12=-(m[index(0,0)]*(m[index(3,3)]*m[index(2,1)]-m[index(2,3)]*m[index(3,1)]) -m[index(0,1)]*(m[index(3,3)]*m[index(2,0)]-m[index(3,0)]*m[index(2,3)]) +m[index(0,3)]*(m[index(3,1)]*m[index(2,0)]-m[index(2,1)]*m[index(3,0)]));
	scalar_t cof13=m[index(0,0)]*(m[index(3,2)]*m[index(2,1)]-m[index(2,2)]*m[index(3,1)]) -m[index(0,1)]*(m[index(3,2)]*m[index(2,0)]-m[index(3,0)]*m[index(2,2)]) +m[index(0,2)]*(m[index(3,1)]*m[index(2,0)]-m[index(2,1)]*m[index(3,0)]);
	
	scalar_t cof20=m[index(0,1)]*(m[index(3,3)]*m[index(1,2)]-m[index(1,3)]*m[index(3,2)]) -m[index(0,2)]*(m[index(3,3)]*m[index(1,1)]-m[index(3,1)]*m[index(1,3)]) +m[index(0,3)]*(m[index(3,2)]*m[index(1,1)]-m[index(3,1)]*m[index(1,2)]);
	scalar_t cof21=-(m[index(0,0)]*(m[index(3,3)]*m[index(1,2)]-m[index(1,3)]*m[index(3,2)]) -m[index(0,2)]*(m[index(3,3)]*m[index(1,0)]-m[index(3,0)]*m[index(1,3)]) +m[index(0,3)]*(m[index(3,2)]*m[index(1,0)]-m[index(3,0)]*m[index(1,2)]));
	scalar_t cof22=m[index(0,0)]*(m[index(3,3)]*m[index(1,1)]-m[index(1,3)]*m[index(3,1)]) -m[index(0,1)]*(m[index(3,3)]*m[index(1,0)]-m[index(3,0)]*m[index(1,3)]) +m[index(0,3)]*(m[index(3,1)]*m[index(1,0)]-m[index(3,0)]*m[index(1,1)]);
	scalar_t cof23=-(m[index(0,0)]*(m[index(3,2)]*m[index(1,1)]-m[index(1,2)]*m[index(3,1)]) -m[index(0,1)]*(m[index(3,2)]*m[index(1,0)]-m[index(3,0)]*m[index(1,2)]) +m[index(0,2)]*(m[index(3,1)]*m[index(1,0)]-m[index(3,0)]*m[index(1,1)]));

	scalar_t cof30=-(m[index(0,1)]*(m[index(2,3)]*m[index(1,2)]-m[index(2,2)]*m[index(1,3)]) -m[index(0,2)]*(m[index(2,3)]*m[index(1,1)]-m[index(2,1)]*m[index(1,3)]) +m[index(0,3)]*(m[index(2,2)]*m[index(1,1)]-m[index(2,1)]*m[index(1,2)]));
	scalar_t cof31=m[index(0,0)]*(m[index(2,3)]*m[index(1,2)]-m[index(2,2)]*m[index(1,3)]) -m[index(0,2)]*(m[index(2,3)]*m[index(1,0)]-m[index(2,0)]*m[index(1,3)]) +m[index(0,3)]*(m[index(2,2)]*m[index(1,0)]-m[index(2,0)]*m[index(1,2)]);
	scalar_t cof32=-(m[index(0,0)]*(m[index(2,3)]*m[index(1,1)]-m[index(2,1)]*m[index(1,3)]) -m[index(0,1)]*(m[index(2,3)]*m[index(1,0)]-m[index(2,0)]*m[index(1,3)]) +m[index(0,3)]*(m[index(2,1)]*m[index(1,0)]-m[index(2,0)]*m[index(1,1)]));
	scalar_t cof33=m[index(0,0)]*(m[index(2,2)]*m[index(1,1)]-m[index(2,1)]*m[index(1,2)]) -m[index(0,1)]*(m[index(2,2)]*m[index(1,0)]-m[index(2,0)]*m[index(1,2)]) +m[index(0,2)]*(m[index(2,1)]*m[index(1,0)]-m[index(2,0)]*m[index(1,1)]);

	scalar_t det=m[index(0,0)]*cof00+m[index(0,1)]*cof01+m[index(0,2)]*cof02+m[index(0,3)]*cof03;
	if(fabs(det)<0.0001)
	{
		#pragma unroll 16 //const time number seemly default unroll? Anyway, here I write down explicitly.
		for(int i=0;i<16;i++)
			// for now, seting inv as zeros
			inv[i]=0.;
		checks[mid]=false;
	}
	else
	{
		inv[index(0,0)]=cof00/det;
		inv[index(0,1)]=cof10/det;
		inv[index(0,2)]=cof20/det;
		inv[index(0,3)]=cof30/det;
		
		inv[index(1,0)]=cof01/det;
		inv[index(1,1)]=cof11/det;
		inv[index(1,2)]=cof21/det;
		inv[index(1,3)]=cof31/det;
		
		inv[index(2,0)]=cof02/det;
		inv[index(2,1)]=cof12/det;
		inv[index(2,2)]=cof22/det;
		inv[index(2,3)]=cof32/det;
		
		inv[index(3,0)]=cof03/det;
		inv[index(3,1)]=cof13/det;
		inv[index(3,2)]=cof23/det;
		inv[index(3,3)]=cof33/det;
		
		checks[mid]=true;
	}
}

template <typename scalar_t>
__global__ void cu4x4MInv_backward(const scalar_t* grads, const scalar_t* invs, scalar_t* outs, int N)
{
	int mid=threadIdx.x+blockIdx.x*blockDim.x;
	if(mid>=N)
		return;
	// const scalar_t* inv=invs+9*mid;
	// const scalar_t* g=grads+9*mid;
	scalar_t* out=outs+9*mid;
	// scalar_t c00=inv[index(0,0)];
	// scalar_t c01=inv[index(0,1)];
	// scalar_t c02=inv[index(0,2)];
	// scalar_t c10=inv[index(1,0)];
	// scalar_t c11=inv[index(1,1)];
	// scalar_t c12=inv[index(1,2)];
	// scalar_t c20=inv[index(2,0)];
	// scalar_t c21=inv[index(2,1)];
	// scalar_t c22=inv[index(2,2)];

	// scalar_t g00=g[index(0,0)];
	// scalar_t g01=g[index(0,1)];
	// scalar_t g02=g[index(0,2)];
	// scalar_t g10=g[index(1,0)];
	// scalar_t g11=g[index(1,1)];
	// scalar_t g12=g[index(1,2)];
	// scalar_t g20=g[index(2,0)];
	// scalar_t g21=g[index(2,1)];
	// scalar_t g22=g[index(2,2)];

	// out[index(0,0)] = -(g00*c00*c00+g01*c00*c01+g02*c00*c02+g10*c10*c00+g11*c10*c01+g12*c10*c02+g20*c20*c00+g21*c20*c01+g22*c20*c02);
	// out[index(0,1)] = -(g00*c00*c10+g01*c00*c11+g02*c00*c12+g10*c10*c10+g11*c10*c11+g12*c10*c12+g20*c20*c10+g21*c20*c11+g22*c20*c12);
	// out[index(0,2)] = -(g00*c00*c20+g01*c00*c21+g02*c00*c22+g10*c10*c20+g11*c10*c21+g12*c10*c22+g20*c20*c20+g21*c20*c21+g22*c20*c22);

	// out[index(1,0)] = -(g00*c01*c00+g01*c01*c01+g02*c01*c02+g10*c11*c00+g11*c11*c01+g12*c11*c02+g20*c21*c00+g21*c21*c01+g22*c21*c02);
	// out[index(1,1)] = -(g00*c01*c10+g01*c01*c11+g02*c01*c12+g10*c11*c10+g11*c11*c11+g12*c11*c12+g20*c21*c10+g21*c21*c11+g22*c21*c12);
	// out[index(1,2)] = -(g00*c01*c20+g01*c01*c21+g02*c01*c22+g10*c11*c20+g11*c11*c21+g12*c11*c22+g20*c21*c20+g21*c21*c21+g22*c21*c22);

	// out[index(2,0)] = -(g00*c02*c00+g01*c02*c01+g02*c02*c02+g10*c12*c00+g11*c12*c01+g12*c12*c02+g20*c22*c00+g21*c22*c01+g22*c22*c02);
	// out[index(2,1)] = -(g00*c02*c10+g01*c02*c11+g02*c02*c12+g10*c12*c10+g11*c12*c11+g12*c12*c12+g20*c22*c10+g21*c22*c11+g22*c22*c12);
	// out[index(2,2)] = -(g00*c02*c20+g01*c02*c21+g02*c02*c22+g10*c12*c20+g11*c12*c21+g12*c12*c22+g20*c22*c20+g21*c22*c21+g22*c22*c22);
	out[index(0,0)] = 1;
	out[index(0,1)] = 2;
	out[index(0,2)] = 3;
	out[index(0,3)] = 4;
	
	out[index(1,0)] = 5;
	out[index(1,1)] = 6;
	out[index(1,2)] = 7;
	out[index(1,3)] = 8;
	
	out[index(2,0)] = 9;
	out[index(2,1)] = 10;
	out[index(2,2)] = 11;
	out[index(2,3)] = 12;
	
	out[index(3,0)] = 13;
	out[index(3,1)] = 14;
	out[index(3,2)] = 15;
	out[index(3,3)] = 16;

}

void M4x4Inv_float(const float* ms, float* invs, bool* checks,int N)
{	
	int threads=1024;
	int blocks=N/threads;
	if(blocks*threads<N)
		blocks+=1;
	cu4x4MInv<float><<<blocks,threads>>>(ms,invs,checks,N);
}

void M4x4Inv_double(const double* ms, double* invs, bool* checks,int N)
{	
	int threads=1024;
	int blocks=N/threads;
	if(blocks*threads<N)
		blocks+=1;
	cu4x4MInv<double><<<blocks,threads>>>(ms,invs,checks,N);
}

void M4x4Inv_backward_float(const float* grads, const float* invs, float* outs,int N)
{	
	int threads=1024;
	int blocks=N/threads;
	if(blocks*threads<N)
		blocks+=1;
	cu4x4MInv_backward<float><<<blocks,threads>>>(grads,invs,outs,N);
}

void M4x4Inv_backward_double(const double* grads, const double* invs, double* outs,int N)
{	
	int threads=1024;
	int blocks=N/threads;
	if(blocks*threads<N)
		blocks+=1;
	cu4x4MInv_backward<double><<<blocks,threads>>>(grads,invs,outs,N);
}